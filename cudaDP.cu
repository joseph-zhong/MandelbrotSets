#include "hip/hip_runtime.h"
// 
// Joseph Zhong
// josephz@cs.washington.edu
// 19 May 2017
// CSE 599I: Final Project
// Instructor Tanner Schmidt
// Exploring Dynamic Parallism in CUDA C with Mandelbrot Sets
// 
// cudaDP.cu
// ---
// 
//  This is the CUDA C implementation with Dynamic Parallelism. 
//

#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <stdbool.h>
#include <time.h>
#include <png.h>
#include <assert.h>

#include "cudaDP.h"
#include "common.h"
#include "defaults.h"
#include "metrics.h"

__host__ void cudaDPMandelbrotSets(int height, int width, int maxIterations, 
    const float radius, const complexNum cMin, const complexNum cMax, const char *filename) {
  const int OUTPUT_SIZE = height * width * sizeof(int);
  int *h_output = (int*) malloc(OUTPUT_SIZE);
  long long int *h_operations = (long long int*) calloc(1, sizeof(long long int));

  int *d_output = NULL; 
  long long int *d_operations = NULL;
  cudaCheck(hipMalloc((void **) &d_output, OUTPUT_SIZE));
  cudaCheck(hipMalloc((void **) &d_operations, sizeof(long long int)));
  cudaCheck(hipMemcpy(d_operations, h_operations, sizeof(long long int), hipMemcpyHostToDevice));

  dim3 gridSize(MIN_SIZE, MIN_SIZE);
  dim3 blockSize(BLOCK_SIZE, DIVIDE_FACTOR);

  clock_t start = clock();

  cudaDPMandelbrotSetsKernel<<<gridSize, blockSize>>>(height, width, maxIterations,
      cMin, cMax, X_POS_DEFAULT, Y_POS_DEFAULT, width / MIN_SIZE, 1, radius,
      d_output, d_operations);
  cudaCheck(hipDeviceSynchronize());
  
  endClock(start);
  
  cudaCheck(hipMemcpy(h_output, d_output, OUTPUT_SIZE, hipMemcpyDeviceToHost));
  cudaCheck(hipMemcpy(h_operations, d_operations, sizeof(long long int), hipMemcpyDeviceToHost));

 	// Free device output and operations.
  hipFree(d_output);
  hipFree(d_operations);
 
	// Write output and operations.
  save_image(filename, h_output, width, height, maxIterations);
  g_operations = *h_operations;
 
  free(h_output);
  free(h_operations);
}




__device__ int commonValue(int v0, int v1, int maxIterations) {
  if (v0 == v1) {
    return v0;
  }
  if (v0 == maxIterations + 1 || v1 == maxIterations + 1) {
    return min(v0, v1);
  }
  return -1;
}

__device__ int calculateBorder(int width, int height, int maxIterations,
    complexNum cMin, complexNum cMax, int x0, int y0, int size, const float radius) {
  int tIdx = threadIdx.y * blockDim.x + threadIdx.x;
  int blockSize = blockDim.x * blockDim.y;
  int value = maxIterations + 1;
  // int value = NEUT_DWELL;
  for (int pixel = tIdx; pixel < size; pixel += blockSize) {
    for (int boundary = 0; boundary < 4; boundary++) {
      int x = boundary % 2 != 0 ? x0 + pixel : (boundary == 0 ? x0 + size - 1 : x0); 
      int y = boundary % 2 == 0 ? y0 + pixel : (boundary == 1 ? y0 + size - 1 : y0);
      value = commonValue(value, calculatePixelValue(width, height, maxIterations, cMin, cMax, x, y, radius), maxIterations);
    }
  }

  __shared__ int s_output[64 * 4];
  int numThreads = min(size, 64 * 4);
  if (tIdx < numThreads) {
    s_output[tIdx] = value;
  }
  __syncthreads();

  // while (numThreads > 1) {
  for(; numThreads > 1; numThreads /= 2) {
    if (tIdx < numThreads / 2) {
      s_output[tIdx] = commonValue(s_output[tIdx], s_output[tIdx + numThreads / 2], maxIterations);
    }
    __syncthreads();
  }
  return s_output[0];
}

__global__ void pixelKernel(int width, int height, int maxIterations, 
    complexNum cMin, complexNum cMax, int x0, int y0, int size, const float radius, int *d_output) {
  int x = threadIdx.x + blockDim.x * blockIdx.x;
  int y = threadIdx.y + blockDim.y * blockIdx.y;

  if (x < size && y < size) {
    x += x0;
    y += y0;
    d_output[y * width + x] = calculatePixelValue(width, height, maxIterations, cMin, cMax, x, y, radius);
    // int outputIndex = CHANNELS * width * y + x * CHANNELS;
    // int pixelValue = calculatePixelValue(width, height, maxIterations, cMin, cMax, x, y, radius);
    // d_output[outputIndex] = (char) pixelValue;
    // d_output[outputIndex + 1] = (char) pixelValue;
    // d_output[outputIndex + 2] = (char) 255;
  }
}

__global__ void fillKernel(int width, int x0, int y0, int size, int value, int *d_output) {
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  if (x < size && y < size) {
    x += x0;
    y += y0;
    d_output[y * width + x] = value;
  }
}

__global__ void cudaDPMandelbrotSetsKernel(int height, int width, int maxIterations,
    complexNum cMin, complexNum cMax, int x0, int y0, int size, int depth, const float radius,
    int *d_output, long long int *d_operations) {

  x0 += size * blockIdx.x;
  y0 += size * blockIdx.y;

  int borderVal = calculateBorder(width, height, maxIterations, cMin, cMax, x0, y0, size, radius); 

  if(threadIdx.x == 0 && threadIdx.y == 0) {
    if (borderVal != -1) {
      dim3 fillBlockSize(64, 4);
      dim3 fillGridSize(divup(size, 64), divup(size, 4));
      fillKernel<<<fillGridSize, fillBlockSize>>>(width, x0, y0, size, borderVal, d_output);
    }
    else if (depth + 1 < MAX_DEPTH && size / 4 > MIN_SIZE) {
      dim3 recurseGridSize(4, 4);
      dim3 recurseBlockSize(blockDim.x, blockDim.y);
      cudaDPMandelbrotSetsKernel<<<recurseGridSize, recurseBlockSize>>>(height, width, maxIterations, 
          cMin, cMax, x0, y0, size / 4, depth + 1, radius, d_output, d_operations); 
    }
    else {
      dim3 pixelGridSize(divup(size, 64), divup(size, 4));
      dim3 pixelBlockSize(64, 4);
      pixelKernel<<<pixelGridSize, pixelBlockSize>>>(width, height, maxIterations,
           cMin, cMax, x0, y0, size, radius, d_output);
    }
  }
}

