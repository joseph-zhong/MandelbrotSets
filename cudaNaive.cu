#include "hip/hip_runtime.h"
// 
// Joseph Zhong
// josephz@cs.washington.edu
// 19 May 2017
// CSE 599I: Final Project
// Instructor Tanner Schmidt
// Exploring Dynamic Parallism in CUDA C with Mandelbrot Sets
// 
// cudaNaive.cu
// ---
// 
//  This is the naive CUDA C implementation without usage of Dynamic
//  Parallelism. 
//

#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <stdbool.h>
#include <time.h>

#include "cudaNaive.h"
#include "common.h"
#include "defaults.h"
#include "metrics.h"

__host__ void cudaNaiveMandelbrotSets(int height, int width, int maxIterations, const float radius, 
    const complexNum cMin, const complexNum cMax, const char *filename) {
	// Host input setup: image.
	const int OUTPUT_SIZE = sizeof(int) * height * width;
	int *h_output = (int*) malloc(OUTPUT_SIZE);

	// Device output setup: image.
	int *d_output;
	cudaCheck(hipMalloc(&d_output, OUTPUT_SIZE));

	// Kernel Size.
	dim3 gridSize(ceil(width / TILE_WIDTH), ceil(height / TILE_WIDTH), 1); 
	dim3 blockSize(TILE_WIDTH, TILE_WIDTH, 1); 

	// Begin timer.
	clock_t start = clock();

	// Launch Kernel.
  cudaNaiveMandelbrotSetsKernel<<<gridSize, blockSize>>>(d_output, width, height, maxIterations, radius, 
      cMin, cMax);

  // Synchronize across threads once completed.
  cudaCheck(hipDeviceSynchronize());

	// Stop timer.
	endClock(start);

  if (filename != NULL) {
    // Copy output.
    cudaCheck(hipMemcpy(h_output, d_output, OUTPUT_SIZE, hipMemcpyDeviceToHost));        

    // Write to output.
    saveImage(filename, h_output, width, height, maxIterations);
  } 

  // Free output.
  hipFree(d_output);
	free(h_output);
}

__global__ void cudaNaiveMandelbrotSetsKernel(int *d_output, 
    int width, int height, int maxIterations, const float radius, 
    complexNum cMin, complexNum cMax) {

  int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
  if (x >= width || y >= height) return;

	int value = calculatePixelValue(width, height, maxIterations,
    cMin, cMax, x, y, radius);
	d_output[y * width + x] = value;
}

