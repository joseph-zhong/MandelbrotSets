#include "hip/hip_runtime.h"
// 
// Joseph Zhong
// josephz@cs.washington.edu
// 19 May 2017
// CSE 599I: Final Project
// Instructor: Tanner Schmidt
// Exploring Dynamic Parallelism in CUDA C with Mandelbrot Sets.
//
// main.cu
// ---
//  This is the main program which launches computer kernels.
//

#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <stdbool.h>
#include <time.h>
#include <string.h>

#include "cudaNaive.h"
#include "common.h"
#include "defaults.h"
#include "metrics.h"
#include "naive.h"
#include "cudaNaive.h"
#include "cudaDP.h"


int main(int argc, char *argv[]) {
  // Default operational values.
  int width;
  int height;
  int x;
  int y;
  int maxIterations;
  char *kernel;
  char *filename;
  FILE *fp;

  parseArgs(argc, argv, &width, &height, &maxIterations, &kernel, &filename);

  if (VERBOSE) {
    printf("\n[main] OPERATING PARAMETERS\n");
    printf("-----------------------------\n");
    printf("\twidth: '%d'\n\theight: '%d'\n\tmaxIterations: '%d'\n\tkernel: '%s'\n\tfilename: '%s'\n\n",
        width, height, maxIterations, kernel, filename);
  }

  // Set filename for output image.
  fp = fopen(filename, "wb");

  // REVIEW josephz: These could be cmdline arguments but in order to
  // standardize the experiments, we will keep these constant for now.
  const float zoom = ZOOM_DEFAULT;
  const float xPos = X_POS_DEFAULT;
  const float yPos = Y_POS_DEFAULT;
  const float radius = RADIUS_DEFAULT;

  double pr, pi;                       //real and imaginary part of the pixel p
  double newRe, newIm, oldRe, oldIm;   //real and imaginary parts of new and old z

  // Write header to ppm file.
  fprintf(fp, "P6\n# Mandelbrot Set. \n%d %d\n255\n", width, height);

  if (strcmp(kernel, NAIVE_HOST) == 0) {
    if (VERBOSE) {
      printf("[main] Running NAIVE_HOST\n\n");
    }
    naiveMandelbrotSets(y, x, height, width, maxIterations, zoom, yPos, xPos, radius, fp);
  }
  if (strcmp(kernel, CUDA_NAIVE) == 0) {
    // Host input setup: image and operations count.
    const int OUTPUT_SIZE = sizeof(char) * height * width * 3;
    char *h_output = (char*) malloc(OUTPUT_SIZE);
    long long int *h_operations = (long long int*) calloc(1, sizeof(long long int));

    // Device output setup: image and operations.
    char *d_output;
    long long int *d_operations;
    cudaCheck(hipMalloc(&d_operations, sizeof(long long int)));
    cudaCheck(hipMalloc(&d_output, OUTPUT_SIZE));
    // cudaCheck(hipMemset(d_operations, 0, OUTPUT_SIZE));

    // Set operations to 0.
    cudaCheck(hipMemcpy(d_operations, h_operations, sizeof(long long int), hipMemcpyHostToDevice));
    
    // Kernel Size.
    dim3 gridSize(ceil(width / TILE_WIDTH), ceil(height / TILE_WIDTH), 1);
    dim3 blockSize(TILE_WIDTH, TILE_WIDTH, 1);

    // Begin timer.
    //struct timespec tstart={0,0};
    clock_t start = clock();

    // Launch Kernel.
    naiveMandelbrotSetsKernel<<<gridSize, blockSize>>>(
        height, width, maxIterations, zoom, yPos, xPos, radius, d_output, d_operations); 
    hipDeviceSynchronize();

    // Stop timer.
    endClock(start);

    // Copy output and operations.
    cudaCheck(hipMemcpy(h_output, d_output, OUTPUT_SIZE, hipMemcpyDeviceToHost));    
    cudaCheck(hipMemcpy(h_operations, d_operations, sizeof(long long int), hipMemcpyDeviceToHost));

    // Free output and operations.
    hipFree(d_output);
    hipFree(d_operations);

    fwrite(h_output, OUTPUT_SIZE, 1, fp);
    g_operations = *h_operations;

    free(h_output);
    free(h_operations);
  }
  
  reportClock();
  reportOperations();
  reportFlops();
  return EXIT_SUCCESS;
}

